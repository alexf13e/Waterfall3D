#include "hip/hip_runtime.h"

#include "SPH.cuh"

#include <iostream>
#include <chrono>
#include <string>

#include "glad/glad.h"

#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "thrust/device_ptr.h"
#include "thrust/sort.h"

#include "glm/gtc/constants.hpp"
#include "CUDAFunctions.cuh"

UniformGrid::~UniformGrid()
{
	destroy();
}

bool UniformGrid::init(const float& smoothingRadius, const float& simRegionSize, const int& numParticles)
{
	initialised = false;

	settings.cellSize = 2 * smoothingRadius;
	settings.dimCells = ceil(simRegionSize / settings.cellSize) + 2;
	settings.numCells = settings.dimCells * settings.dimCells * settings.dimCells;

	hipMalloc((void**)&data.d_cellIDs, sizeof(int) * numParticles);
	hipMalloc((void**)&data.d_particleIDs, sizeof(int) * numParticles);
	hipMalloc((void**)&data.d_cellStarts, sizeof(int) * settings.numCells);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "Uniform grid init: error during cuda memory allocation: " << hipGetErrorName(err) << std::endl;
		return false;
	}

	initialised = true;

	return true;
}

void UniformGrid::destroy()
{
	hipFree(data.d_cellIDs);
	hipFree(data.d_particleIDs);
	hipFree(data.d_cellStarts);

	initialised = false;
}

void UniformGrid::update(const SPHSimulationData& simData, const SPHConfiguration& simSettings)
{
	if (!initialised) return;

	int blockSize = glm::min(simSettings.numParticles, 1024);
	int numBlocks = (simSettings.numParticles - 1) / blockSize + 1;

	//update which cells contain which particles
#if ENABLE_TIMING_SPH
	std::chrono::steady_clock::time_point t1 = std::chrono::high_resolution_clock::now();
	CUDAKernels::UGUpdateCellParticles<<<numBlocks, blockSize>>>(simData, simSettings, this->data, this->settings);
	hipDeviceSynchronize();
	std::chrono::steady_clock::time_point t2 = std::chrono::high_resolution_clock::now();
	float dt = (t2 - t1).count() * 1e-9;
	std::cout << dt << ",";
#else
	CUDAKernels::UGUpdateCellParticles<<<numBlocks, blockSize>>>(simData, simSettings, this->data, this->settings);
#endif

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "CUDA error in UGUpdateCellParticles: " << hipGetErrorName(err) << std::endl;
	}

	//cellIDs[i] contains the index of the cell which particle with index i is in
	//particleIDs[i] contains i, just as a list of the all particle IDs, which will be sorted to maintain the pairing
	//with cellIDs when cellIDs is sorted to be ascending


	//sort particles by which cell they are in
	//cellID is key, particleID is the value
	//both keys and values will be sorted
#if ENABLE_TIMING_SPH
	t1 = std::chrono::high_resolution_clock::now();
	thrust::sort_by_key(thrust::device_ptr<int>(data.d_cellIDs),
		thrust::device_ptr<int>(data.d_cellIDs + simSettings.numParticles),
		thrust::device_ptr<int>(data.d_particleIDs));
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	dt = (t2 - t1).count() * 1e-9;
	std::cout << dt << ",";
#else
	thrust::sort_by_key(thrust::device_ptr<int>(data.d_cellIDs),
		thrust::device_ptr<int>(data.d_cellIDs + simSettings.numParticles),
		thrust::device_ptr<int>(data.d_particleIDs));
#endif

	//update cellStarts
	//need to be initialised as having no cells, then ones which do have cells will be overwritten
#if ENABLE_TIMING_SPH
	t1 = std::chrono::high_resolution_clock::now();
	hipMemset(data.d_cellStarts, -1, sizeof(int) * settings.numCells);
	CUDAKernels::UGUpdateCellStarts<<<numBlocks, blockSize>>>(simSettings, data);
	hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
	dt = (t2 - t1).count() * 1e-9;
	std::cout << dt << ",";
#else
	hipMemset(data.d_cellStarts, -1, sizeof(int) * settings.numCells);
	CUDAKernels::UGUpdateCellStarts<<<numBlocks, blockSize>>>(simSettings, data);
#endif
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "CUDA error in UGUpdateCellStarts: " << hipGetErrorName(err) << std::endl;
	}

	//cellStarts[i] contains the index for the first particle in particleIDs which is in cell with id i
	//if no particles in cell i, then it contains -1
}


SPHSolver::~SPHSolver()
{
	destroy();
}

bool SPHSolver::init(const SPHConfiguration& settings, const float& simRegionSize)
{
	if (initialised)
	{
		std::cerr << "attempted to initialise solver which was already initialised" << std::endl;
		return false;
	}

	simSettings = settings;
	
	//placeholder arrays initialised to 0
	glm::vec3* emptyDataVec3 = new glm::vec3[simSettings.numParticles]();
	float* emptyDataFloat = new float[simSettings.numParticles]();

	//triangular array of distances between pairs of particles, size follows equation 0.5n(n - 1)
	simSettings.numParticleDistances = 0.5 * simSettings.numParticles * (simSettings.numParticles - 1);
	float* emptyDistances = new float[simSettings.numParticleDistances]();

	//create simulation data on GPU
	//particle positions need to be accessible by opengl for rendering, requiring them to be created in openGL and mapped to cuda
	glGenVertexArrays(1, &simData.gl_positionsVAO);
	glBindVertexArray(simData.gl_positionsVAO);
	glGenBuffers(1, &simData.gl_positionsVBO);
	glBindBuffer(GL_ARRAY_BUFFER, simData.gl_positionsVBO);
	glBufferData(GL_ARRAY_BUFFER, simSettings.numParticles * sizeof(glm::vec3), emptyDataVec3, GL_DYNAMIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(glm::vec3), 0);
	glEnableVertexAttribArray(0);

	glGenVertexArrays(1, &simData.gl_predictedPositionsVAO);
	glBindVertexArray(simData.gl_predictedPositionsVAO);
	glGenBuffers(1, &simData.gl_predictedPositionsVBO);
	glBindBuffer(GL_ARRAY_BUFFER, simData.gl_predictedPositionsVBO);
	glBufferData(GL_ARRAY_BUFFER, simSettings.numParticles * sizeof(glm::vec3), emptyDataVec3, GL_DYNAMIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(glm::vec3), 0);
	glEnableVertexAttribArray(0);

	glBindVertexArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);


	//tell cuda where the buffer is
	hipGraphicsGLRegisterBuffer(&simData.cuda_positionsBuffer, simData.gl_positionsVBO, cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsGLRegisterBuffer(&simData.cuda_predictedPositionsBuffer, simData.gl_predictedPositionsVBO, cudaGraphicsMapFlagsWriteDiscard);

	//other data can be simply stored only for access by cuda
	hipMalloc((void**)&simData.d_velocities, sizeof(glm::vec3) * simSettings.numParticles);
	hipMalloc((void**)&simData.d_accelerations, sizeof(glm::vec3) * simSettings.numParticles);
	hipMalloc((void**)&simData.d_densities, sizeof(float) * simSettings.numParticles);
	hipMalloc((void**)&simData.d_distances, sizeof(float) * simSettings.numParticleDistances);

	hipMemcpy(simData.d_velocities, emptyDataVec3, sizeof(glm::vec3) * simSettings.numParticles, hipMemcpyHostToDevice);
	hipMemcpy(simData.d_accelerations, emptyDataVec3, sizeof(glm::vec3) * simSettings.numParticles, hipMemcpyHostToDevice);
	hipMemcpy(simData.d_densities, emptyDataFloat, sizeof(float) * simSettings.numParticles, hipMemcpyHostToDevice);
	hipMemcpy(simData.d_distances, emptyDistances, sizeof(float) * simSettings.numParticleDistances, hipMemcpyHostToDevice);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "sph solver init: error during cuda memory allocation: " << hipGetErrorName(err) << std::endl;
		return false;
	}

#if USE_UNIFORM_GRID
	uniformGrid = new UniformGrid();
	if (uniformGrid->init(simSettings.smoothingRadius, simRegionSize, simSettings.numParticles) == false) return false;
#endif

	timeElapsed = 0.0f;
	initialised = true;

	return true;
}

void SPHSolver::destroy()
{
	glDeleteBuffers(1, &simData.gl_positionsVBO);
	glDeleteBuffers(1, &simData.gl_predictedPositionsVBO);
	glDeleteVertexArrays(1, &simData.gl_positionsVAO);
	glDeleteVertexArrays(1, &simData.gl_predictedPositionsVAO);

	hipFree(simData.d_velocities);
	hipFree(simData.d_accelerations);
	hipFree(simData.d_densities);
	hipFree(simData.d_distances);

	//uniformGrid deleted automatically when solver class is
	
	initialised = false;
}

bool SPHSolver::update(int iterations)
{
	if (!initialised)
	{
		std::cerr << "attempted to update before intialisation" << std::endl;
		return false;
	}

	int blockSize = glm::min(simSettings.numParticles, 1024);
	int numBlocks = (simSettings.numParticles - 1) / blockSize + 1;

	mapCudaResources(); //to use particle position data which is shared with opengl
	while (iterations > 0)
	{
#if ENABLE_TIMING_SPH
		std::chrono::steady_clock::time_point t1 = std::chrono::high_resolution_clock::now();
		CUDAKernels::updatePredictedParticlePositions << <numBlocks, blockSize >> > (simData, simSettings);
		hipDeviceSynchronize();
		std::chrono::steady_clock::time_point t2 = std::chrono::high_resolution_clock::now();
		float dt = (t2 - t1).count() * 1e-9;
		//std::cout << "Kernel time: updatePredictedParticlePositions: " << std::to_string(dt * 1e3) << "ms" << std::endl;
		std::cout << std::to_string(dt) << ",";
#else
		CUDAKernels::updatePredictedParticlePositions << <numBlocks, blockSize >> > (simData, simSettings);
#endif
		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "CUDA error in updatePredictedParticlePositions: " << hipGetErrorName(err) << std::endl;
		}

#if ENABLE_TIMING_SPH
		t1 = std::chrono::high_resolution_clock::now();
		CUDAKernels::calculateInterParticleValues << <numBlocks, blockSize >> > (simData, simSettings);
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		dt = (t2 - t1).count() * 1e-9;
		//std::cout << "Kernel time: calculateInterParticleValues: " << std::to_string(dt * 1e3) << "ms" << std::endl;
		std::cout << std::to_string(dt) << ",";
#else
		CUDAKernels::calculateInterParticleValues << <numBlocks, blockSize >> > (simData, simSettings);
#endif
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "CUDA error in calculateInterParticleValues: " << hipGetErrorName(err) << std::endl;
		}

#if ENABLE_TIMING_SPH
		t1 = std::chrono::high_resolution_clock::now();
		CUDAKernels::processTimeStep << <numBlocks, blockSize >> > (simData, simSettings);
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		dt = (t2 - t1).count() * 1e-9;
		//std::cout << "Kernel time: processTimeStep: " << std::to_string(dt * 1e3) << "ms" << std::endl;
		std::cout << std::to_string(dt) << ",";
#else
		CUDAKernels::processTimeStep << <numBlocks, blockSize >> > (simData, simSettings);
#endif
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "CUDA error in processTimeStep: " << hipGetErrorName(err) << std::endl;
		}

		iterations--;
	}
	unmapCudaResources(); //must be unmapped before being used by opengl (or else)


	timeElapsed += simSettings.timeStep;

	return true;
}

bool SPHSolver::UGUpdate(int iterations)
{
	if (!initialised)
	{
		std::cerr << "attempted to update before intialisation" << std::endl;
		return false;
	}

	int blockSize = glm::min(simSettings.numParticles, 1024);
	int numBlocks = (simSettings.numParticles - 1) / blockSize + 1;

	const UniformGridSettings& ugSettings = uniformGrid->getSettings();
	const UniformGridData& ugData = uniformGrid->getData();

	mapCudaResources(); //to use particle position data which is shared with opengl
	while (iterations > 0)
	{
#if ENABLE_TIMING_SPH
		std::chrono::steady_clock::time_point t1 = std::chrono::high_resolution_clock::now();
		CUDAKernels::updatePredictedParticlePositions << <numBlocks, blockSize >> > (simData, simSettings);
		hipDeviceSynchronize();
		std::chrono::steady_clock::time_point t2 = std::chrono::high_resolution_clock::now();
		float dt = (t2 - t1).count() * 1e-9;
		//std::cout << "Kernel time: updatePredictedParticlePositions: " << std::to_string(dt * 1e3) << "ms" << std::endl;
		std::cout << std::to_string(dt) << ",";
#else
		CUDAKernels::updatePredictedParticlePositions << <numBlocks, blockSize >> > (simData, simSettings);
#endif
		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "CUDA error in updatePredictedParticlePositions: " << hipGetErrorName(err) << std::endl;
		}

		//predicted positions should be set before being used by uniform grid
		uniformGrid->update(simData, simSettings);

#if ENABLE_TIMING_SPH
		t1 = std::chrono::high_resolution_clock::now();
		CUDAKernels::UGCalculateInterParticleValues << <numBlocks, blockSize >> > (simData, simSettings, ugData, ugSettings);
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		dt = (t2 - t1).count() * 1e-9;
		//std::cout << "Kernel time: calculateInterParticleValues: " << std::to_string(dt * 1e3) << "ms" << std::endl;
		std::cout << std::to_string(dt) << ",";
#else
		CUDAKernels::UGCalculateInterParticleValues << <numBlocks, blockSize >> > (simData, simSettings, ugData, ugSettings);
#endif
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "CUDA error in calculateInterParticleValues: " << hipGetErrorName(err) << std::endl;
		}

#if ENABLE_TIMING_SPH
		t1 = std::chrono::high_resolution_clock::now();
		CUDAKernels::UGProcessTimeStep << <numBlocks, blockSize >> > (simData, simSettings, ugData, ugSettings);
		hipDeviceSynchronize();
		t2 = std::chrono::high_resolution_clock::now();
		dt = (t2 - t1).count() * 1e-9;
		//std::cout << "Kernel time: processTimeStep: " << std::to_string(dt * 1e3) << "ms" << std::endl;
		std::cout << std::to_string(dt) << ",";
#else
		CUDAKernels::UGProcessTimeStep << <numBlocks, blockSize >> > (simData, simSettings, ugData, ugSettings);
#endif
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cerr << "CUDA error in processTimeStep: " << hipGetErrorName(err) << std::endl;
		}

		iterations--;
	}
	unmapCudaResources(); //must be unmapped before being used by opengl (or else)
	

	timeElapsed += simSettings.timeStep;

	return true;
}

void SPHSolver::setInitialParticlePositions(const float spacing)
{
	if (!initialised)
	{
		std::cerr << "attempted to set initial particle positions before intialisation" << std::endl;
		return;
	}

	//generate positions in as close to a square around the origin as possible, filling space in order x, z, then y
	//find minimum width, height and depth to fit the number of particles into a cube
	int target = simSettings.numParticles;
	int w = 1;
	int d = 1;
	int h = 1;

	while (w * d * h < target)
	{
		w++; if (w * d * h >= target) break;
		d++; if (w * d * h >= target) break;
		h++;
	}

	int blockSize = glm::min(simSettings.numParticles, 1024);
	int numBlocks = (simSettings.numParticles - 1) / blockSize + 1;

	mapCudaResources();
		CUDAKernels::setInitialParticlePositions<<<numBlocks, blockSize>>>(simData.d_positions,
			simSettings.numParticles, w, h, d, spacing);
	unmapCudaResources();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "CUDA error in setInitialParticlePositions: " << hipGetErrorName(err) << std::endl;
	}
}

void SPHSolver::setWorldBoundaries(Boundary* boundaries, int count)
{
	if (simData.h_worldBoundaries != nullptr)
	{
		delete[] simData.h_worldBoundaries;
		hipFree(simData.d_worldBoundaries);
	}

	simData.h_worldBoundaries = new Boundary[count];
	simData.worldBoundaryCount = count;
	
	memcpy(simData.h_worldBoundaries, boundaries, count * sizeof(Boundary));

	hipMalloc((void**)&simData.d_worldBoundaries, count * sizeof(Boundary));
	hipMemcpy(simData.d_worldBoundaries, simData.h_worldBoundaries, count * sizeof(Boundary), hipMemcpyHostToDevice);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cerr << "CUDA error in setWorldBoundaries: " << hipGetErrorName(err) << std::endl;
	}
}

//void SPHSolver::userInteractParticles(const glm::vec2& attractionPoint, const float attractionRadius,
//	const float attractionVelocity)
//{
//	int blockSize = glm::min(simSettings.numParticles, 1024);
//	int numBlocks = (simSettings.numParticles - 1) / blockSize + 1;
//
//	mapCudaResources();
//		CUDAKernels::userInteractParticles<<<numBlocks, blockSize>>>(simData, simSettings.numParticles, attractionPoint,
//			attractionRadius, attractionVelocity);
//	unmapCudaResources();
//
//	hipError_t err = hipGetLastError();
//	if (err != hipSuccess)
//	{
//		std::cerr << "CUDA error in userInteractParticles: " << hipGetErrorName(err) << std::endl;
//	}
//}

void SPHSolver::mapCudaResources()
{
	size_t _numbytes;
	hipGraphicsMapResources(1, &simData.cuda_positionsBuffer);
	hipGraphicsMapResources(1, &simData.cuda_predictedPositionsBuffer);
	hipGraphicsResourceGetMappedPointer((void**)&simData.d_positions, &_numbytes, simData.cuda_positionsBuffer);
	hipGraphicsResourceGetMappedPointer((void**)&simData.d_predictedPositions, &_numbytes, simData.cuda_predictedPositionsBuffer);
}

void SPHSolver::unmapCudaResources()
{
	hipGraphicsUnmapResources(1, &simData.cuda_positionsBuffer, 0);
	hipGraphicsUnmapResources(1, &simData.cuda_predictedPositionsBuffer, 0);

	//pointers are no longer valid after unmapping, set to null just in case they are used by mistake
	simData.d_positions = nullptr;
	simData.d_predictedPositions = nullptr;
}